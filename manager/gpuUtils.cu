// C i/o  for printf and others
#include <stdio.h>
#include <vector>

// CUDA specifics

#include <hip/hip_runtime.h>

#ifdef CUDA_ERROR_CHECK
#include <helper_functions.h>
#include <helper_cuda.h>
#endif

// Define the macros
#define CudaSafeCall(err) __cudaSafeCall(err, __FILE__, __LINE__)
#define CudaCheckError()  __cudaCheckError(__FILE__, __LINE__)

/// KS: Need it for shared memory, there is way to use dynamic shared memory but I am lazy right now
#define _BlockSize_ 1024

// CUDA_ERROR_CHECK is now defined in the makefile instead
//#define CUDA_ERROR_CHECK

// **************************************************
//             ERROR CHECKING ROUTINES
// Also exist in helper_cuda.h
// **************************************************

// **************************************************
/// @brief Check for a safe call on GPU
inline void __cudaSafeCall( hipError_t err, const char *file, const int line ) {
// **************************************************
#ifdef CUDA_ERROR_CHECK
  if (cudaSuccess != err) {
    fprintf(stderr, "cudaSafeCall() failed at %s:%i : %s\n", file, line, cudaGetErrorString(err));
    exit(-1);
  }
#endif
  return;
}

// **************************************************
/// @brief Check if there's been an error
inline void __cudaCheckError( const char *file, const int line ) {
// **************************************************
#ifdef CUDA_ERROR_CHECK
  cudaError err = cudaGetLastError();
  if (cudaSuccess != err) {
    fprintf(stderr, "cudaCheckError() failed at %s:%i : %s\n", file, line, cudaGetErrorString(err));
    exit(-1);
  }

  // More careful checking. However, this will affect performance.
  // Comment away if needed.
  err = cudaDeviceSynchronize();
  if (cudaSuccess != err) {
    fprintf(stderr, "cudaCheckError() with sync failed at %s:%i : %s\n", file, line, cudaGetErrorString(err));
    exit(-1);
  }
#endif
  return;
}

// *******************************************
//              Utils
// *******************************************

// *******************************************
/// @brief KS: Get some fancy info about VRAM usage
inline void checkGpuMem() {
// *******************************************

  float free_m, total_m,used_m;
  size_t free_t, total_t;

  hipMemGetInfo(&free_t, &total_t);
  CudaCheckError();

  free_m = (uint)free_t/1048576.0;
  total_m = (uint)total_t/1048576.0;
  used_m = total_m - free_m;

  printf("  Memory free %f MB, total memory %f MB, memory used %f MB\n", free_m, total_m, used_m);
}

// *******************************************
/// @brief KS: Get some fancy info about GPU
inline void PrintNdevices() {
// *******************************************

  int nDevices;
  hipGetDeviceCount(&nDevices);
  CudaCheckError();

  if (nDevices == 0) {
    printf("No CUDA devices found");
    throw;
  }

  printf("  Found %i GPUs, currently I only support one GPU\n", nDevices);
}


// *******************************************
/// @brief KS: Completely clean GPU, this is time consuming and may lead to unexpected behaviour.
inline void ResetDevice() {
// *******************************************

  hipDeviceReset();
  CudaCheckError();
}


// *******************************************
/// @brief Only useful if using multiple GPU
inline void SetDevice(const int deviceId) {
// *******************************************

  // Check if the device ID is valid
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  if (deviceId < 0 || deviceId >= deviceCount) {
    printf("Invalid device ID: %i \n", deviceId);
    throw;
  }

  hipSetDevice(deviceId);
  CudaCheckError();
  printf("GPU device set to ID: %i \n", deviceId);

}

// *******************************************
/// @brief Get number of GPU threads for currently used GPU
inline void GetNumGPUThreads(const int Device = 0) {
// *******************************************

  int deviceCount;
  hipGetDeviceCount(&deviceCount);

  if (deviceCount == 0) {
    printf("No CUDA devices found");
    throw;
  }

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, Device);

  // Define the number of threads per block
  int nThreadsBlocks = (deviceProp.multiProcessorCount * deviceProp.maxThreadsPerMultiProcessor);

  printf("Currently used GPU has : %i threads \n", nThreadsBlocks);
}
